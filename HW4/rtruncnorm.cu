#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>
//for boolean functionality
#include <stdbool.h>

extern "C"
{

__global__ void 
rtruncnorm_kernel(float *vals, int n, 
                  float *mu, float *sigma, 
                  float *lo, float *hi,
                  int mu_len, int sigma_len,
                  int lo_len, int hi_len,
                  int max_tries, 
                  int rng_a, int rng_b, int rng_c)
{
    // Usual block/thread indexing...
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    int idx = myblock * blocksize + subthread;

    //mkae sure index is not overrunning the number of threads
    if (idx < N) {
        printf("WARNING! Exceeded number of threads. Did not sample.");
        return;
    }

    // Setup the RNG:
    hiprandState rng_state;
    hiprand_init(rng_a + idx*rng_b, rng_c, 0, &rng_state);

    //Rejection Sampling:
    int rejected = 0;
    int iter_count = 0;
    while (rejected == 0 && iter_count < max_tries) {
        iter_count = iter_count + 1;
        vals[idx] = mu[idx] + sigma[idx] * hiprand_normal(&rng_state); 
        if (vals[idx] > lo[idx] && vals[idx] <= hi[idx]) {
            rejected = 1;
        }
    }
    return;
}

} // END extern "C"

