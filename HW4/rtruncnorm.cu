#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>
#include <math.h>
//for boolean functionality
#include <stdbool.h>

extern "C"
{

__global__ void 
rtruncnorm_kernel(float *vals, int n, 
                  float *mu, float *sigma, 
                  float *lo, float *hi,
                  int mu_len, int sigma_len,
                  int lo_len, int hi_len,
                  int max_tries, 
                  int rng_a, int rng_b, int rng_c)
{
    /* Usual block/thread indexing. Note this code, according to Prof. Baines,
       is only robust when the grid and block dims have the structure like;
            (x, 1,1)
       */
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    int idx = myblock * blocksize + subthread;

    //make sure index is not overrunning the number of threads
    if (idx < n) {

	    // Setup the RNG:
	    hiprandState rng_state;
	    hiprand_init(rng_a + idx*rng_b, rng_c, 0, &rng_state);

	    /*
	       Rejection Sampling:
	       (i) First try two-sided truncation naive approach. It works for cases 
where: |high - low| >> 0. which will be true for this homework 
assignment. This code is not robust to really tricky two-sided 
truncation. because the probability of sampling from that region 
is high. 
(ii) When that fails for the one-sided tail regions, that have a small 
probability of being sampled, then apply the Robert approach. 
	     */

	    int accepted = 0;
	    int iter_count = 0;
	    while (accepted == 0 && iter_count < max_tries) {
		    iter_count = iter_count + 1;
		    vals[idx] = mu[idx] + sigma[idx] * hiprand_normal(&rng_state); 
		    //accepted or not?
		    if (vals[idx] > lo[idx] && vals[idx] <= hi[idx]) {
			    accepted = 1;
			    return;
		    }
	    }

	    /*If it never accepted, then for this assignment we can assume that
	      we have a case of heavy right or left truncation where we are 
	      trying to sample from only one of the tails. 
	     */
	    if (accepted == 0) {

		    /*right truncation requires adaptation because the Robert-rejection 
		      sampling for one-sided truncation defaults to left truncation*/

		    //indicate whether it is right truncated to flip the sign of the 
		    //sampled value if right_trunc = 1.
		    int right_trunc;

		    float mu_tmp = mu[idx];
		    float lo_tmp = lo[idx];

		    if (hi[idx] <  mu_tmp) {
			    right_trunc = 1;
			    mu_tmp = -1 * mu_tmp;
			    lo_tmp = -1 * hi[idx];
		    } else {
			    //left truncation
			    right_trunc = 0;
		    }

		    //see Appendix A below
		    int mu_minus = ( lo_tmp  - mu_tmp ) / sigma[idx];

		    /****************************************/
		    /* left truncation, right tail sampling*/

		    /*step 0: set the optimal rate parameter for the exponential 
		      distribution*/
		    float alpha =  ( mu_minus + sqrtf(mu_minus*mu_minus + 4) ) / 2;

		    while (accepted == 0) {
			    /*step 1:generate: z ~ Expo(\alpha, \mu_minus)
			      by the inv-cdf transform (since z is continous)
			     */
			    float z = mu_minus - log (hiprand_uniform(&rng_state)) / alpha; 

			    /*step 2: compute ratio h(z) / ( M * g(z) ) */
			    float psi;
			    float offset1 = alpha - z;
			    float offset2;
			    if (mu_minus < alpha) { 
				    psi  = exp( -0.5 * offset1*offset1 );
			    } else {
				    offset2 = mu_minus - alpha;
				    psi = exp( -0.5 * ( offset1*offset1 + offset2*offset2 ) );
			    }

			    //accepted the sample
			    if (hiprand_uniform(&rng_state) <= psi) {
				    if (right_trunc == 1) {
					    vals[idx] = -1 * (mu_tmp + sigma[idx]*z);
				    } else {
					    vals[idx] = mu_tmp + sigma[idx]*z;
				    }
				    accepted = 1;
				    return;
			    }
		    } // END while loop
	    } // END Robert rejection-sampling.
    }
    return;
} // END rtruncnorm_kernel

} // END extern "C"

/*Appendix A*/
/*Need to adjust the truncation boundary of X, which is
          lo[idx] to the truncation boundary of a standard normal (Z):
          
          Std Normal (left truncation):
          Z ~ N(\mu = 0, \sigma = 1, low_z = \mu_minus, hi_z = \Inf)
          
          Standardized location-scale relation:
          Z = (X - \mu_x) / \sigma_x 
          X = \mu_x + \sigma_x * Z
          X ~ N(\mu = \mu_x, 
                \sigma = \sigma_x, 
                lo[idx] = \sigma_x*\mu_minus + \mu_x,
                \Inf)

          Then:
          \mu_minus = ( lo[idx] - \mu_x ) / sigma_x;

*/


